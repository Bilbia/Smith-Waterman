#include "hip/hip_runtime.h"
#include <algorithm>  
#include <iostream>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/copy.h>
#include <cstdlib>
#include <iomanip>
#include <tuple>


using namespace std;

struct calcMax{
    int diag, cima;
    char c;
    calcMax(int c_) : c(c_){};
    __host__ __device__
    int operator()(const thrust::tuple<char, int, int>& a){

        if(c == thrust::get<0>(a)){
            diagonal = thrust::get<1>(a) + 2;
        }else{
            diagonal = thrust::get<1>(a) - 1;
        }
        cima = thrust::get<2>(a) - 1;
        if ( diagonal > 0 && diagonal > cima ){
            return diagonal;
        }else if( cima > 0 && cima > diagonal){
            return cima;
        }else if(diagonal > 0 && diagonal == cima ){
            return cima;
        }else{
            return 0;
        }
    }
};

struct comp{
    __host__ __device__
    int operator()(const int& x, const int& y){
        int left = x - 1;
        int cell = y;

        if (cell > left && cell > 0){
            return cell;
        }else if(left > cell && left > 0){
            return left;
        }else if(cell == left  && cell > 0){
            return cell;
        }else{
            return 0;
        }
    }
};

int main(){
    int m, n; //tamanho da primeira e segunda sequencia
    string seq1, seq2;
    cin >> m >> n >> seq1 >> seq2;

    seq1 = '_' + seq1;
    seq2 = '_' + seq2;

    int max = -(m + n);
    int temp, lenMax;
    int k = 1;

    thrust::device_vector<char> seq1GPU(m+1);
    thrust::device_vector<char> seq2GPU(n+1);
    thrust::device_vector<int> H1(n+1);
    thrust::device_vector<int> H2(n+1);
    thrust::fill(H2.begin(), H2.end(), 0);

    for(int i = 0; i < m + 1; i++) {
        seq1GPU[i] = seq1[i];
    }
    for(int i = 0; i < n + 1; i++) {
        seq2GPU[i] = seq2[i];
    }

    if(m > n) {
        lenMax = n;
    }
    else {
        lenMax = m;
    }

    m++;
    n++;
    
    while(lenMax >= k && lenMax > 1) {
        for(int i = 0; i <= m - lenMax; i++){
            cout << i << endl;
            for(int j = 0; j <= n - lenMax; j++){
                thrust::fill(H1.begin(), H1.end(), 0);

                for(int l = 0; l < lenMax + 1; l++){
                    thrust::transform(thrust::make_zip_iterator(thrust::make_tuple(seq2GPU.begin() + 1 + j, H1.begin(), H1.begin() + 1)),
                                        thrust::make_zip_iterator(thrust::make_tuple(seq2GPU.end() + 1 + j + lenMax, H1.begin() + lenMax, H1.begin() + 1 + lenMax)),
                                        H2.begin() + 1,
                                        calcMax(seq1GPU[l]));

                    thrust::inclusive_scan(H2.begin(), H2.begin() + lenMax, H1.begin(), comp());

                    temp = thrust::reduce(H1.begin(), H1.begin() + lenMax, 0, thrust::maximum<int>());
                    
                    if(temp > max){
                        max = temp;
                    }
                }
            }
        }
        k = (max / 2) + 1;
        lenMax --;
        cout << "MAIOR SCORE: "<< max << endl;
   
    }
    return 0;
}

